#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define cudaCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            printf("[ERROR] Failed to run stmt %s\n", #stmt);                       \
            printf("[ERROR] Got CUDA error %s\n", hipGetErrorString(err));    \
            return;                                                        \
        }                                                                     \
    } while(0)


#define UNROLL_TILE_WIDTH 16
__global__ void unroll_conv_forward_kernel(float *y, const float *x, const float* k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.
    Function paramter definitions:
    y - output
    x - input
    k - kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    */
    __shared__ float subTileM[UNROLL_TILE_WIDTH][UNROLL_TILE_WIDTH];
    __shared__ float subTileN[UNROLL_TILE_WIDTH][UNROLL_TILE_WIDTH];


    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int b = blockIdx.z;
    int W_unroll = H_out * W_out;
    int H_unroll = C * K * K;
    int numAColumns = H_unroll;
    int m_upper = (numAColumns + UNROLL_TILE_WIDTH - 1) / UNROLL_TILE_WIDTH;

    int row = by * UNROLL_TILE_WIDTH + ty;
    int col = bx * UNROLL_TILE_WIDTH + tx;

    float result = 0;

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = y4d(0,0,0,0)
    // y4d(0,0,0,0) = a

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    
    for (int m = 0; m < m_upper; m++) {
        // Fill the shared memory
        int m_col = (m * UNROLL_TILE_WIDTH + tx);
        if (m_col < numAColumns && row < M) {
            subTileM[ty][tx] = k4d(row, m_col/(K*K), (m_col%(K*K))/K, (m_col%(K*K))%K);
        }
        else {
            subTileM[ty][tx] = 0;
        }
        int n_row = (m * UNROLL_TILE_WIDTH + ty);
        int x_c = n_row / (K * K);
        int x_base_h = col / W_out;
        int x_base_w = col % W_out;
        int x_p = (n_row % (K * K)) / K;
        int x_q = (n_row % (K * K)) % K;
        if (n_row < numAColumns && col < W_unroll) {
            subTileN[ty][tx] = x4d(b, x_c, x_base_h + x_p, x_base_w + x_q);
        }
        else {
            subTileN[ty][tx] = 0;
        }
        __syncthreads();
        for (int k = 0; k < UNROLL_TILE_WIDTH; k++)
        {
            result += subTileM[ty][k] * subTileN[k][tx];
        }
        __syncthreads();
    }

    if (row < M && col < W_unroll) {
        y4d(b, row, col / W_out, col % W_out) = result;
    }

#undef y4d
#undef x4d
#undef k4d
}

__constant__ float mask[4096]; // >= 16*4*7*7
#define SHARE_TILE_WIDTH 16
// We make sure that each kernel handle TILE_WIDTH*TILE_WIDTH*c output
// From mp4
__global__ void share_conv_forward_kernel(float *y, const float *x, const int B, const int M, const int C, const int H, const int W, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.
    Function paramter definitions:
    y - output
    x - input
    k - kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    */
    extern __shared__ float SM[];
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    const int SM_WIDTH = SHARE_TILE_WIDTH + K - 1;
    const int grid_subh = ceil((float)H_out / (float)SHARE_TILE_WIDTH);
    const int grid_subw = ceil((float)W_out / (float)SHARE_TILE_WIDTH);
    const int b = blockIdx.x;
    const int m = blockIdx.y;
    const int z = blockIdx.z;
    const int tile_h = z / grid_subw;
    const int tile_w = z % grid_subw;
    const int base_h = tile_h * SHARE_TILE_WIDTH;
    const int base_w = tile_w * SHARE_TILE_WIDTH;
    const int h = base_h + threadIdx.y;
    const int w = base_w + threadIdx.x;
    float result = 0;

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = y4d(0,0,0,0)
    // y4d(0,0,0,0) = a

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    for(int c = 0; c < C; c++) {
        // First load our share memory
        for (int iter_h = h; iter_h < base_h + SM_WIDTH; iter_h += SHARE_TILE_WIDTH) {
            for (int iter_w = w; iter_w < base_w + SM_WIDTH; iter_w += SHARE_TILE_WIDTH) {
                const int sm_h = iter_h - base_h;
                const int sm_w = iter_w - base_w;
                const int sm_idx = sm_h * SM_WIDTH + sm_w;
                if (b < B && iter_h < H && iter_w < W) {
                    SM[sm_idx] = x4d(b, c, iter_h, iter_w);
                }
                else {
                    SM[sm_idx] = 0.0f;
                }
            }
        }
            
        __syncthreads();
        // Compute!
        #pragma unroll
        for (int p = 0; p < K; p++) {
            #pragma unroll
            for (int q = 0; q < K; q++) {
                result += SM[(threadIdx.y+p) * SM_WIDTH + (threadIdx.x+q)]  * k4d(m, c, p, q);
            }
        }
        __syncthreads();
    }

    if(b < B && m < M && h < H_out && w < W_out) {
        y4d(b, m, h, w) = result;
    }

#undef y4d
#undef x4d
#undef k4d
}

__host__ void GPUInterface::conv_forward_gpu(float *host_y, const float *host_x, const float *host_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Declare relevant device pointers
    float *device_y;
    float *device_x;
    float *device_k;

    // Init size
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    int y_size = B * M * H_out * W_out;
    int x_size = B * C * H * W;
    int k_size = M * C * K * K;
    int y_bytes = y_size * sizeof(float);
    int x_bytes = x_size * sizeof(float);
    int k_bytes = k_size * sizeof(float);
    int W_unroll = H_out * W_out;
    int H_unroll = C * K * K;

    printf("B: %d, M: %d, C: %d, H: %d, W: %d, K: %d, H_out: %d, W_out: %d, W_unroll: %d, H_unroll: %d\n", B, M, C, H, W, K, H_out, W_out, W_unroll, H_unroll);

    // Allocate memory and copy over the relevant data structures to the GPU
    cudaCheck(hipMalloc((void **)&device_y, y_bytes));
    cudaCheck(hipMalloc((void **)&device_x, x_bytes));
    cudaCheck(hipMalloc((void **)&device_k, k_bytes));

    cudaCheck(hipMemcpy((void *)device_x, (void *)host_x, x_bytes, hipMemcpyHostToDevice));

    cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(mask), host_k, k_bytes));
    cudaCheck(hipMemcpy((void *)device_k, (void *)host_k, k_bytes, hipMemcpyHostToDevice));

    // std::cout << "Done Allocate memory\n";

    // // Set up unroll matrix
    // float *device_x_unroll;
    // int W_unroll = H_out * W_out;
    // int H_unroll = C * K * K;
    // int x_unroll_bytes = W_unroll * H_unroll * sizeof(float);
    // cudaCheck(hipMalloc((void **)&device_x_unroll, x_unroll_bytes));

    // dim3 dimUnrollBlock(BLOCK_SIZE);
    // dim3 dimUnrollGrid(ceil((float)(C*H_out*W_out) / (float)BLOCK_SIZE));

    // dim3 dimMulBlock(TILE_WIDTH, TILE_WIDTH);
    // dim3 dimMulGrid(ceil((float)W_unroll / TILE_WIDTH), ceil((float)M / TILE_WIDTH));

    // // Set the kernel dimensions and call the kernel
    // for (int b = 0; b < B; b++) {
    //     unroll_kernel<<<dimUnrollGrid, dimUnrollBlock>>>(device_x, device_x_unroll, b, C, H, W, K, W_unroll, W_out);
    //     matrixMultiplyShared<<<dimMulGrid, dimMulBlock>>>(device_k, device_x_unroll, device_y + b * M * H_out * W_out, H_unroll, M, W_unroll);
    // }

    // std::cout << "Done calling kernel\n";

    if(M == 4) {
        const int grid_subh = ceil((float)H_out / (float)SHARE_TILE_WIDTH);
        const int grid_subw = ceil((float)W_out / (float)SHARE_TILE_WIDTH);
        const int SM_WIDTH = SHARE_TILE_WIDTH + K - 1;
        const int SM_SIZE = SM_WIDTH * SM_WIDTH * sizeof(float);

        dim3 dimBlock(SHARE_TILE_WIDTH, SHARE_TILE_WIDTH);
        dim3 dimGrid(B, M, grid_subh * grid_subw);

        share_conv_forward_kernel<<<dimGrid, dimBlock, SM_SIZE>>>(device_y, device_x, B, M, C, H, W, K);
    } else {
        dim3 dimBlock(UNROLL_TILE_WIDTH, UNROLL_TILE_WIDTH, 1);
        dim3 dimGrid(ceil((float)W_unroll / UNROLL_TILE_WIDTH), ceil((float)M / UNROLL_TILE_WIDTH), B);

        unroll_conv_forward_kernel<<<dimGrid, dimBlock>>>(device_y, device_x, device_k, B, M, C, H, W, K);
    }

    // Copy the output back to host
    cudaCheck(hipMemcpy((void *)host_y, (void *)device_y, y_bytes, hipMemcpyDeviceToHost));

    // std::cout << "Done copy back\n";

    // Free device memory
    cudaCheck(hipFree(device_y));
    cudaCheck(hipFree(device_x));
    cudaCheck(hipFree(device_k));
    // cudaCheck(hipFree(device_x_unroll));

    // std::cout << "Done free memory\n";

    // Useful snippet for error checking
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
}

__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}